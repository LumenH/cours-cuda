#include "hip/hip_runtime.h"
#include "MontecarloMulti.h"

#include "Device.h"
#include <hiprand/hiprand_kernel.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

extern __global__ void montecarlo(int* ptrDevResult, hiprandState* ptrTabDevGenerator, int nbFlechette, float a, float b, float M);
extern __global__ void setup_kernel_rand(hiprandState* ptrTabDevGenerator, int deviceId);
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

MontecarloMulti::MontecarloMulti(const Grid& grid,int nbFlechette){
    this->sizeOctetGM = sizeof(int);
    this->sizeOctetSM = sizeof(int) * grid.db.x;
    this->sizeOctetGeneratorGM = sizeof(hiprandState) * grid.threadCounts();

    this->result = 0;
    this->nbFlechetteDessous = 0;
    this->nbFlechette = nbFlechette;

    this->ptrDevResult = NULL;
    this->ptrTabDevGenerator = NULL;

    this->dg = grid.dg;
    this->db = grid.db;

    this->a = (float) -1;
    this->b = (float) 1;
    this->M = 1.0f;

    Device::malloc(&ptrDevResult, sizeOctetGM);
    Device::memclear(ptrDevResult, sizeOctetGM);

    Device::malloc(&ptrTabDevGenerator, sizeOctetGeneratorGM);
    Device::memclear(ptrTabDevGenerator, sizeOctetGeneratorGM);

    int deviceId = Device::getDeviceId();

    setup_kernel_rand<<<dg, db>>>(ptrTabDevGenerator, deviceId);//a check pour ce qui est entre les triples chevrons
}

MontecarloMulti::~MontecarloMulti(void){
    Device::free(ptrDevResult);
    Device::free(ptrTabDevGenerator);
}

void MontecarloMulti::run(){
    montecarlo<<<dg, db, sizeOctetSM>>>(ptrDevResult, ptrTabDevGenerator, nbFlechette, a, b, M);
    Device::memcpyDToH(&nbFlechetteDessous, ptrDevResult, sizeOctetGM);
    float delta = fabsf(b-a);
    float area = M*delta;
    float ratio = (float)nbFlechetteDessous/(float)nbFlechette;
    this->result = 2*area*ratio;
}

float MontecarloMulti::getResult(){
    return this->result;
}

int MontecarloMulti::getNbFlechette(){
    return this->nbFlechetteDessous;
}


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

