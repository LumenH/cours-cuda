#include "MontecarloMultiGPU.h"
#include "Device.h"
#include "MontecarloMulti.h"

MontecarloMultiGPU::MontecarloMultiGPU(const Grid& grid, const int nbFlechetteTotale){
    this->nbDevice = Device::getDeviceCount();
    this->grid = grid;
    this->nbFlechetteTotale = nbFlechetteTotale;
    this->result = 100;
}

void MontecarloMultiGPU::runMulti(){
    int nbFlechetteGPU = nbFlechetteTotale/this->nbDevice;

    long sumTotal = 0;

    #pragma omp parallel for reduction(+:sumTotal)
    for(int idDevice = 0; idDevice < this->nbDevice; idDevice++){
	hipSetDevice(idDevice);
	MontecarloMulti montecarlomulti(this->grid, nbFlechetteGPU);
	montecarlomulti.run();
	sumTotal += montecarlomulti.getNbFlechette();
    }
 //il manque les formules de finition
}

float MontecarloMultiGPU::getResult(){
    return this->result;
}

