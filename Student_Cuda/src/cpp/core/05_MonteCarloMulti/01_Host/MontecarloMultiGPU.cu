#include "hip/hip_runtime.h"
#include "MontecarloMultiGPU.h"
#include "Device.h"
#include "MontecarloMulti.h"

MontecarloMultiGPU::MontecarloMultiGPU(const Grid& grid, const int nbFlechetteTotale){
    this->nbDevice = Device::getDeviceCount();
    this->grid = grid;
    this->nbFlechetteTotale = nbFlechetteTotale;
    this->result = 0;
}

void MontecarloMultiGPU::runMulti(){
    int nbFlechetteGPU = nbFlechetteTotale/this->nbDevice;

    long sumTotal = 0;

    #pragma omp parallel for reduction(+:sumTotal)
    for(int idDevice = 0; idDevice < this->nbDevice; idDevice++){
	hipSetDevice(idDevice);
	MontecarloMulti montecarlomulti(this->grid, nbFlechetteGPU);
	montecarlomulti.run();
	sumTotal += montecarlomulti.getNbFlechette();
    }

    //this->result = sumTotal;
    float delta = fabsf(-1.0f-1.0f);
    float area = 1.0f*delta;
    float ratio = (float)sumTotal/(float)nbFlechetteTotale;
    this->result = 2*area*ratio;
}

float MontecarloMultiGPU::getResult(){
    return this->result;
}

