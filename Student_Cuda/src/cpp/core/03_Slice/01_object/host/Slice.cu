#include "hip/hip_runtime.h"
#include "Slice.h"

#include "Device.h"

extern __global__ void slice(float* ptrDevRseult, int nbSlice);

Slice::Slice(const Grid& grid,float* ptrResult, int n){
    this->sizeOctet = sizeof(float);
    this->ptrDevResult = NULL;

    {
	Device::malloc(&ptrDevResult, sizeOctet);
	Device::memclear(ptrDevResult, sizeOctet);
	Device::memcpyHToD(ptrDevResult, ptrResult, sizeOctet);

    }

    {
    this->dg = grid.dg;
    this->db = grid.db;
    }

    this->sizeOctetSM = sizeof(float)*db.x;//taille de float * nbThread
}

Slice::~Slice(){
    Device::free(ptrDevResult);
}

void Slice::run(){
    slice<<<dg, db, sizeOctetSM>>>(ptrDevResult, n);
}
