#include "hip/hip_runtime.h"
#include "Slice.h"

#include "Device.h"

extern __global__ void slice(float* ptrDevRseult, int nbSlice);

Slice::Slice(const Grid& grid,float* ptrResult, int nbSlice){

    this->sizeOctetGM = sizeof(float);
    this->sizeOctetSM = sizeof(float) * grid.db.x;
    this->ptrDevResult = NULL;
    this->ptrResult = ptrResult;
    this->nbSlice = nbSlice;

    Device::malloc(&ptrDevResult, sizeOctetGM);
    Device::memclear(ptrDevResult, sizeOctetGM);

    this->dg = grid.dg;
    this->db = grid.db;
}

Slice::~Slice(){
    Device::free(ptrDevResult);
}

void Slice::run(){
    slice<<<dg, db, sizeOctetSM>>>(ptrDevResult, nbSlice);
    Device::memcpyDToH(ptrResult, ptrDevResult, sizeOctetGM);
}
