#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include "reductionADD.h"


__global__ void slice(float* ptrDevResult, int nbSlice);
static __device__ void reductionIntraThread(float* tabSM, int nbSlice);

__global__ void slice(float* ptrDevResult, int nbSlice){
    __shared__ extern float tabSM[];
    reductionIntraThread(tabSM, nbSlice);
    __syncthreads();//barrière pour les threads d'un même bloc
    reductionADD<float>(tabSM, ptrDevResult);//méthode générique
}

static __device__ void reductionIntraThread(float* tabSM, int nbSlice){
    const int TID_Local = threadIdx.x;
    tabSM[TID_Local] = threadIdx.x + blockIdx.x*blockDim.x; //On remplit de 1.
}
