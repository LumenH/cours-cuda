#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include "reductionADD.h"


__global__ void slice(float* ptrDevResult, int nbSlice);
static __device__ void reductionIntraThread(float* tabSM, int nbSlice);
__device__ float fpi(float x);

__global__ void slice(float* ptrDevResult, int nbSlice){
    __shared__ extern float tabSM[];
    reductionIntraThread(tabSM, nbSlice);
    __syncthreads();//barrière pour les threads d'un même bloc
    reductionADD<float>(tabSM, ptrDevResult);//méthode générique
}

static __device__ void reductionIntraThread(float* tabSM, int nbSlice){
    //const int TID_Local = threadIdx.x;

    const int NB_THREAD=Indice1D::nbThread();
    const int TID=Indice1D::tid();
    const int TID_LOCAL=Indice1D::tidLocal();
    const float DX = 1.f / (float) nbSlice;

    int s = TID;
    float tmp = 0;
    while(s < nbSlice)
    {
    	tmp += fpi(s * DX);
    	s += NB_THREAD;
    }

    tabSM[TID_LOCAL] = tmp * DX;
}

__device__ float fpi(float x)
{
    return 4.f / (1.f + x * x);
}
