#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <limits.h>
#include <Indice1D.h>
#include "reductionADD.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__global__ void setup_kernel_rand(hiprandState* ptrTabDevGenerator, int deviceId);
static __device__ void reductionIntraThread(int* tabSM, int nbFlechette, hiprandState* ptrTabDevGenerator);
__global__ void montecarlo(int* ptrDevResult, hiprandState* ptrTabDevGenerator, int nbFlechette);
__device__ int work(float xAlea, float yAlea);
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void montecarlo(int* ptrDevResult, hiprandState* ptrTabDevGenerator, int nbFlechette){
    __shared__ extern int tabSM[];
    reductionIntraThread(tabSM, nbFlechette, ptrTabDevGenerator);
    __syncthreads();//barrière pour les threads d'un même bloc
    reductionADD<int>(tabSM, ptrDevResult);//méthode générique
}

static __device__ void reductionIntraThread(int* tabSM, int nbFlechette, hiprandState* ptrTabDevGenerator){
    const int TID_Local = Indice1D::tidLocal();
    //tabSM[TID_Local] = 1;
    const int TID = Indice1D::tid();

    hiprandState localGenerator = ptrTabDevGenerator[TID];

    float xAlea, yAlea;
    float y;
    int n = 0;

    for(long i = 1; i <= nbFlechette; i++){
	xAlea = hiprand_uniform(&localGenerator);
	yAlea = hiprand_uniform(&localGenerator);
	n += work(xAlea, yAlea);
	/*y = work(xAlea, yAlea);

	if(y < yAlea){
	    n++;
	}*/
    }

    tabSM[TID_Local]=n;
    ptrTabDevGenerator[TID] = localGenerator;
}

__device__ int work(float xAlea, float yAlea){
    float fPi = (1.0 /(1.0 +xAlea*xAlea))*4;
    return (int) (fPi >= yAlea);
    //return (4.0/(1.0 + xAlea * xAlea));
}

__global__ void setup_kernel_rand(hiprandState* ptrTabDevGenerator, int deviceId)
{
// Customisation du generator:
// Proposition, au lecteur de faire mieux !
// Contrainte : Doit etre différent d'un GPU à l'autre
// Contrainte : Doit etre différent d’un thread à l’autre
    const int TID = Indice1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;
    hiprand_init(seed, sequenceNumber, offset, &ptrTabDevGenerator[TID]);
}


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

