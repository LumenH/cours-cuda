#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <limits.h>
#include <Indice1D.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void setup_kernel_rand(hiprandState* ptrTabDevGenerator, int deviceId)
{
// Customisation du generator:
// Proposition, au lecteur de faire mieux !
// Contrainte : Doit etre différent d'un GPU à l'autre
// Contrainte : Doit etre différent d’un thread à l’autre
    const int TID = Indice1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;
    hiprand_init(seed, sequenceNumber, offset, &ptrTabDevGenerator[tid]);
}


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

