#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <limits.h>
#include <Indice1D.h>
#include "reductionADD.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__global__ void setup_kernel_rand(hiprandState* ptrTabDevGenerator, int deviceId);
static __device__ void reductionIntraThread(int* tabSM, int nbFlechette);
__global__ void montecarlo(int* ptrDevResult, hiprandState* ptrTabDevGenerator, int nbFlechette);
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void montecarlo(int* ptrDevResult, hiprandState* ptrTabDevGenerator, int nbFlechette){
    __shared__ extern int tabSM[];
    reductionIntraThread(tabSM, nbFlechette);
    __syncthreads();//barrière pour les threads d'un même bloc
    reductionADD<int>(tabSM, ptrDevResult);//méthode générique
}

static __device__ void reductionIntraThread(int* tabSM, int nbFlechette){
    //const int TID_Local = threadIdx.x;
    //tabSM[TID_Local] = 1;
}

__global__ void setup_kernel_rand(hiprandState* ptrTabDevGenerator, int deviceId)
{
// Customisation du generator:
// Proposition, au lecteur de faire mieux !
// Contrainte : Doit etre différent d'un GPU à l'autre
// Contrainte : Doit etre différent d’un thread à l’autre
    const int TID = Indice1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;
    hiprand_init(seed, sequenceNumber, offset, &ptrTabDevGenerator[TID]);
}


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

