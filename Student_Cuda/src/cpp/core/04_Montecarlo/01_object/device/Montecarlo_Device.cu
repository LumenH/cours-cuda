#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <limits.h>
#include <Indice1D.h>
#include "reductionADD.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__global__ void setup_kernel_rand(hiprandState* ptrTabDevGenerator, int deviceId);
static __device__ void reductionIntraThread(int* tabSM, int nbFlechette, hiprandState* ptrTabDevGenerator, float a, float b, float M);
__global__ void montecarlo(int* ptrDevResult, hiprandState* ptrTabDevGenerator, int nbFlechette, float a, float b, float M);
__device__ int work(float xAlea, float yAlea);
__device__ float toAB(float x, float a, float b);
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void montecarlo(int* ptrDevResult, hiprandState* ptrTabDevGenerator, int nbFlechette, float a, float b, float M){
    __shared__ extern int tabSM[];
    reductionIntraThread(tabSM, nbFlechette, ptrTabDevGenerator, a, b, M);
    __syncthreads();//barrière pour les threads d'un même bloc
    reductionADD<int>(tabSM, ptrDevResult);//méthode générique
}

__device__ float toAB(float x, float a, float b){
    return (b - a) / 1.0f * x + a;
}

__device__ float f(float x){
    return sqrtf(1.0f - x * x);
}

static __device__ void reductionIntraThread(int* tabSM, int nbFlechette, hiprandState* ptrTabDevGenerator, float a, float b, float M){
    const int TID_Local = Indice1D::tidLocal();
    //tabSM[TID_Local] = 1;
    const int TID = Indice1D::tid();

    hiprandState localGenerator = ptrTabDevGenerator[TID];

    float xAlea, yAlea;
    //float y;
    int n = 0;

    for(long i = 1; i <= nbFlechette; i++){
	xAlea = toAB(hiprand_uniform(&localGenerator), a, b);
	yAlea = toAB(hiprand_uniform(&localGenerator), a, b);
	if(yAlea < f(xAlea)){
	    n++;
	}
    }

    tabSM[TID_Local]=n;
    ptrTabDevGenerator[TID] = localGenerator;
}


__device__ int work(float xAlea, float yAlea){
    float fPi = (1.0 /(1.0 +xAlea*xAlea))*4;
    return (int) (fPi >= yAlea);
    //return (4.0/(1.0 + xAlea * xAlea));
}

__global__ void setup_kernel_rand(hiprandState* ptrTabDevGenerator, int deviceId)
{
// Customisation du generator:
// Proposition, au lecteur de faire mieux !
// Contrainte : Doit etre différent d'un GPU à l'autre
// Contrainte : Doit etre différent d’un thread à l’autre
    const int TID = Indice1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;
    hiprand_init(seed, sequenceNumber, offset, &ptrTabDevGenerator[TID]);
}


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

