#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <limits.h>
#include <Indice1D.h>
#include "reductionADD.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__global__ void setup_kernel_rand(hiprandState* ptrTabDevGenerator, int deviceId);
static __device__ void reductionIntraThread(int* tabSM, int nbFlechette, hiprandState* ptrTabDevGenerator, float a, float b, float M);
__global__ void montecarlo(int* ptrDevResult, hiprandState* ptrTabDevGenerator, int nbFlechette, float a, float b, float M);
__device__ float toAB(float x, float a, float b);
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void montecarlo(int* ptrDevResult, hiprandState* ptrTabDevGenerator, int nbFlechette, float a, float b, float M){
    __shared__ extern int tabSM[];
    reductionIntraThread(tabSM, nbFlechette, ptrTabDevGenerator, a, b, M);
    __syncthreads();//barrière pour les threads d'un même bloc
    reductionADD<int>(tabSM, ptrDevResult);//méthode générique
}

__device__ float toAB(float x, float a, float b){
    return (b - a) / 1.0f * x + a;
}

__device__ float f(float x){
    return sqrtf(1.0f - x * x);
}

static __device__ void reductionIntraThread(int* tabSM, int nbFlechette, hiprandState* ptrTabDevGenerator, float a, float b, float M){
    const int TID_Local = Indice1D::tidLocal();
    //tabSM[TID_Local] = 1;
    const int TID = Indice1D::tid();
    const int NB_THREAD = Indice1D::nbThread();

    hiprandState localGenerator = ptrTabDevGenerator[TID];

    float xAlea, yAlea;
    int n = 0;
    int s = TID;

    //for(long i = 1; i <= nbFlechette; i++){
    while(s < nbFlechette){
	xAlea = toAB(hiprand_uniform(&localGenerator), a, b);
	yAlea = toAB(hiprand_uniform(&localGenerator), 0.0f, M);
	if(yAlea < f(xAlea)){
	    n++;
	}
	s += NB_THREAD;
    }

    tabSM[TID_Local]=n;
    //ptrTabDevGenerator[TID] = localGenerator;
}

__global__ void setup_kernel_rand(hiprandState* ptrTabDevGenerator, int deviceId)
{
// Customisation du generator:
// Proposition, au lecteur de faire mieux !
// Contrainte : Doit etre différent d'un GPU à l'autre
// Contrainte : Doit etre différent d’un thread à l’autre
    const int TID = Indice1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;
    hiprand_init(seed, sequenceNumber, offset, &ptrTabDevGenerator[TID]);
}


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

