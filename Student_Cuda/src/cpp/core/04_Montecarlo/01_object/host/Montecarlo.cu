#include "hip/hip_runtime.h"
#include "Montecarlo.h"
#include "Device.h"
#include <hiprand/hiprand_kernel.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

extern __global__ void montecarlo(int* ptrDevResult, hiprandState* ptrTabDevGenerator, int nbFlechette, float a, float b, float M)
extern __global__ void setup_kernel_rand(hiprandState* ptrTabDevGenerator, int deviceId);
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Montecarlo::Montecarlo(const Grid& grid,int nbFlechette){
    this->sizeOctetGM = sizeof(int);
    this->sizeOctetSM = sizeof(int) * grid.db.x;
    this->sizeOctetGeneratorGM = sizeof(hiprandState) * grid.threadCounts();

    this->result = 0;
    this->nbFlechetteDessous = 0;
    this->nbFlechette = nbFlechette;

    this->ptrDevResult = NULL;
    this->ptrTabDevGenerator = NULL;

    this->dg = grid.dg;
    this->db = grid.db;

    Device::malloc(&ptrDevResult, sizeOctetGM);
    Device::memclear(ptrDevResult, sizeOctetGM);

    Device::malloc(&ptrTabDevGenerator, sizeOctetGeneratorGM);
    Device::memclear(ptrTabDevGenerator, sizeOctetGeneratorGM);

    int deviceId = Device::getDeviceId();

    setup_kernel_rand<<<dg, db>>>(ptrTabDevGenerator, deviceId);//a check pour ce qui est entre les triples chevrons
}

Montecarlo::~Montecarlo(void){
    Device::free(ptrDevResult);
    Device::free(ptrTabDevGenerator);
}

void Montecarlo::run(){
    montecarlo<<<dg, db, sizeOctetSM>>>(ptrDevResult, ptrTabDevGenerator, nbFlechette, a, b, M);
    Device::memcpyDToH(&nbFlechetteDessous, ptrDevResult, sizeOctetGM);
    this->result = (float) nbFlechetteDessous;
}

float Montecarlo::getResult(int m){
    return (this->result/(float)this->nbFlechette)*(float)m;
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

