#include "hip/hip_runtime.h"
#include "Montecarlo.h"
#include "Device.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

extern __global__ void montecarlo(float* ptrDevResult, hiprandState* ptrTabDevGenerator, int nbFlechette);
extern __global__ void setup_kernel_rand(hiprandState* ptrTabDevGenerator, int deviceId);
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Montecarlo::Montecarlo(const &Grid grid, float* ptrResult, int nbFlechette){
    this->sizeOctetGM = sizeof(float);
    this->sizeOctetSM = sizeof(float) * nbFlechette;//grid.db.x;
    this->sizeOctetGeneratorGM = sizeof(hiprandState) * grid.threadCounts();

    this->ptrDevResult = NULL;
    this->ptrResult = ptrResult;
    this->nbFlechette = nbFlechette;
    this->dg = grid.dg;
    this->db = grid.db;

    Device::malloc(&ptrDevResult, sizeOctetGM);
    Device::memclear(ptrDevResult, sizeOctetGM);

    Device::malloc(&ptrTabDevGenerator, sizeOctetGeneratorGM);
    Device::memclear(ptrTabDevGenerator, sizeOctetGeneratorGM);

    int deviceId = Device::getDeviceId();

    setup_kernel_rand<<<dg, db>>>(ptrTabDevGenerator, deviceId);//a check pour ce qui est entre les triples chevrons


}

Montecarlo::~Montecarlo(void){
    Device::free(ptrDevResult);
    Device::free(ptrTabDevGenerator);
}

void Montecarlo::run(){
    montecarlo<<<dg, db, sizeOctetSM>>(ptrDevResult, ptrTabDevGenerator, nbFlechette);
    Device::memcpyHToD(ptrDevResult, ptrResult, sizeOctetGM);
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

