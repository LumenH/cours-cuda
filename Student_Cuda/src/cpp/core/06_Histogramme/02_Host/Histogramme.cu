#include "hip/hip_runtime.h"
#include <iostream>

#include "Histogramme.h"
#include "Device.h"

const int SIZE_TAB_SM = 256;

extern __global__ void histogramme(int* ptrTabData, int sizeTab, int* ptrDevResult);

Histogramme::Histogramme(const Grid& grid, int sizeTab){

    this->datacreator = new DataCreator(sizeTab);
    this->ptrTabData = this->datacreator->getTabData();
    this->n = this->datacreator->getLength();

    this->ptrTabResult = new int[SIZE_TAB_SM];

    this->sizeOctet= sizeof(int)*this->n;
    this->sizeOctetResult = sizeof(int)*SIZE_TAB_SM;

    this->dg = grid.dg;
    this->db = grid.db;
    this->sizeTab = sizeTab;

    DataCreator dataCreator(SIZE_TAB_SM-1);

    Device::malloc(&ptrDevData, sizeOctet);
    Device::memclear(ptrDevData, sizeOctet);

    Device::malloc(&ptrDevResult, sizeOctetResult);
    Device::memclear(ptrDevResult, sizeOctetResult);

    Device::memcpyHToD(this->ptrDevData, this->ptrTabData, this->sizeOctet);

}

int* Histogramme::run(void){
    histogramme<<<dg, db, sizeOctetResult>>>(this->ptrDevData, this->n, this->ptrDevResult);
    Device::synchronize();
    Device::memcpyDToH(ptrTabResult, ptrDevResult, sizeOctetResult);
    return ptrTabResult;
}

Histogramme::~Histogramme(void){
    Device::free(ptrDevResult);
    Device::free(ptrDevData);
}
