#include "hip/hip_runtime.h"
#include <stdio.h>

#include "Indice2D.h"
#include "cudaTools.h"

__global__ void histogramme(int* ptrTabData, int sizeTab, int* ptrDevResult);
static __device__ void reductionIntraThread(int* ptrTabData, int sizeTab, int *tabSM);
__device__ void reductionInterBlock(int* TAB_SM, int* ptrGM);

__global__ void histogramme(int* ptrTabData, int sizeTab, int* ptrDevResult){
    extern __shared__ int tabSM[];

    if (Indice2D::tidLocal() == 0){
	for (int i = 0; i < 256; i++){
	    tabSM[i] = 0;
	}
    }

    __syncthreads();

    reductionIntraThread(ptrTabData, sizeTab, tabSM);
    __syncthreads();
    reductionInterBlock(tabSM, ptrDevResult);

}

__device__ void reductionInterBlock(int* tabSM, int* tabGM){
    if (Indice2D::tidLocal() == 0){
	for (int i = 0; i < 256; i++){
	    atomicAdd(&tabGM[i], tabSM[i]);
    	}
    }
}

__device__ void reductionIntraThread(int* ptrTabData, int sizeTab, int* tabSM){
    const int NB_THREAD = Indice2D::nbThread();
    const int TID_LOCAL = Indice2D::tidLocalBlock();
    const int TID = Indice2D::tid();

    int s = TID;

    while(s < sizeTab){
	atomicAdd(&tabSM[ptrTabData[s]], 1);
	s+= NB_THREAD;
    }
}
