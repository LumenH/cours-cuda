#include "hip/hip_runtime.h"
#include "cudaTools.h"
#include "Indice2D.h"

#include "IndiceTools_GPU.h"
#include "RaytracingMath.h"

using namespace gpu;

__global__ void raytracing(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphere, int nbSphere){
    RaytracingMath rm = RaytracingMath(ptrDevTabSphere, nbSphere);

    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();
    const int WH = w*h;

    int s = TID;
    int i, j;

    while(s < WH){
	IndiceTools::toIJ(s, w, &i,  &j);
	rm.colorIJ(&ptrDevPixels[s], i, j, t);
	s += NB_THREAD;
    }
}
