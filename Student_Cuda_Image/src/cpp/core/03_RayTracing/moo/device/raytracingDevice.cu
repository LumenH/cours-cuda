#include "hip/hip_runtime.h"
#include "cudaTools.h"
#include "Indice2D.h"

#include "IndiceTools_GPU.h"
#include "Sphere.h"
#include "RaytracingMath.h"

using namespace gpu;

__global__ void raytracing(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphere, int nbSphere){
    RaytracingMath rm = RaytracingMath(ptrDevTabSphere, nbSphere);

    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();
    const int WH = w*h;

    int s = TID;
    //int n = nbSphere;
    int i, j;
    //float hcarree = 0;

    while(s < WH){
	IndiceTools::toIJ(s, w, &i,  &j);
	rm.colorIJ(&ptrDevPixels[s], i, j, t);
	s += NB_THREAD;
    }
}
