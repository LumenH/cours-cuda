#include "hip/hip_runtime.h"
#include "cudaTools.h"
#include "Indice2D.h"

#include "IndiceTools_GPU.h"
#include "Sphere.h"

using namespace gpu;

__global__ void raytracing(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevtabSphere, int nbSphere){
    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();
    const int WH = w*h;

    int s = TID;
    //int n = nbSphere;
    int i, j;
    //float hcarree = 0;

    while(s < WH){
	IndiceTools::toIJ(s, w, &i,  &j);
	//todo workpixel

	s += NB_THREAD;
    }
}
