#include "hip/hip_runtime.h"
#include "cudaTools.h"
#include "Indice2D.h"

#include "IndiceTools_GPU.h"
#include "RaytracingMath.h"

#include "length_cm.h"

using namespace gpu;
__constant__ Sphere TAB_CM[LENGTH_CM];

__device__ void work(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, int nbSphere, const int NB_THREAD, const int TID, uint w, uint h, float t){
    RaytracingMath rm = RaytracingMath(ptrDevTabSphere, nbSphere);
    const int WH = w*h;
    int s = TID;
    int i, j;

    while(s < WH){
	IndiceTools::toIJ(s, w, &i,  &j);
	rm.colorIJ(&ptrDevPixels[s], i, j, t);
	s += NB_THREAD;
    }
}

__global__ void raytracingGM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphere, int nbSphere){

    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();

    work(ptrDevPixels, ptrDevTabSphere, nbSphere, NB_THREAD, TID, w, h, t);
}

__global__ void raytracingCM(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSphere){

    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();

    work(ptrDevPixels, TAB_CM, nbSphere, NB_THREAD, TID, w, h, t);
}

__global__ void raytracingSM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphere, int nbSphere){
    extern __shared__ Sphere ptrDevTabSphereSM[];

    //Attention changement TID et NB_THREAD
    const int NB_THREAD_Local = Indice2D::nbThreadLocal();
    const int TID_Local = Indice2D::tidLocal();

    int s = TID_Local;
    while(s < nbSphere){
	ptrDevTabSphereSM[s] = ptrDevTabSphere[s];
	s++;
    }

    work(ptrDevPixels, ptrDevTabSphereSM, nbSphere, NB_THREAD_Local, TID_Local, w, h, t);
}


__host__ void uploadGPU(Sphere* tabValue){
    size_t size= LENGTH_CM *sizeof(Sphere);
    int offset=0;
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(TAB_CM), tabValue, size, offset, hipMemcpyHostToDevice));
}

