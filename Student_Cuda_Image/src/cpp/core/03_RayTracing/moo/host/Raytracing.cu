#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Raytracing.h"
#include "SphereCreator.h"
#include "Device.h"

#include "length_cm.h"

using std::cout;
using std::endl;


//Global memory part
extern __global__ void raytracing(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphere, int nbSphere);
extern __host__ void uploadGPU(Sphere* tabValue);
extern __global__ void raytracingGM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphere, int nbSphere);
extern __global__ void raytracingCM(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSphere);
extern __global__ void raytracingSM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphere, int nbSphere);
Raytracing::Raytracing(const Grid& grid, uint w, uint h, float dt, int nbSphere): Animable_I<uchar4>(grid, w, h, "Raytracing_CUDA"){
    this->dt = dt;
    this->t = 0;
    this->nbSphere = nbSphere;

    SphereCreator sphereCreator(nbSphere, w, h);
    Sphere* ptrTabSphere = sphereCreator.getTabSphere();

    this->sizeOctet = sizeof(Sphere)*nbSphere;

    //Transfert to Global Memory
    toGM(ptrTabSphere);

    //Transfert to CM
    uploadGPU(ptrTabSphere);
}

void Raytracing::toGM(Sphere* ptrTabSphere){
   Device::malloc(&ptrDevTabSphere, sizeOctet);
   Device::memclear(ptrDevTabSphere, sizeOctet);
   Device::memcpyHToD(ptrDevTabSphere, ptrTabSphere, sizeOctet);

}

void Raytracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath){
    static int i=0;
    //Device::lastCudaError("Before raytracing");
    if (i%3 == 0){
	raytracingGM<<<dg, db>>>(ptrDevPixels, w, h, t, ptrDevTabSphere, nbSphere);
    }
    else if (i%3 == 1){
	raytracingCM<<<dg, db>>>(ptrDevPixels, w, h, t, nbSphere);
    }
    else if (i%3 == 2){
	raytracingSM<<<dg, db, sizeOctet>>>(ptrDevPixels, w, h, t, ptrDevTabSphere, nbSphere);
    }
    //Device::lastCudaError("After raytracing");
    i++;

    //Ajouter barriere synchro implicite
}



Raytracing::~Raytracing(){
    Device::free(ptrDevTabSphere);
}

void Raytracing::animationStep(){
    t += dt;
}


