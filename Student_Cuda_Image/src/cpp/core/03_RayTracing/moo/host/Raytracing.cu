#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Raytracing.h"
#include "SphereCreator.h"
#include "Device.h"

using std::cout;
using std::endl;

extern __global__ void raytracing(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphere, int nbSphere);

Raytracing::Raytracing(const Grid& grid, uint w, uint h, float dt, int nbSphere): Animable_I<uchar4>(grid, w, h, "Raytracing_CUDA"){
    this->dt = dt;
    this->t = 0;
    this->nbSphere = nbSphere;

    SphereCreator sphereCreator(nbSphere, w, h);
    Sphere* ptrTabSphere = sphereCreator.getTabSphere();

    this->sizeOctet = sizeof(Sphere)*nbSphere;

    //Transfert to Global Memory
    toGM(ptrTabSphere);

    //Transfert to CM
    //toCM(ptrTabSphere);
}

void Raytracing::toGM(Sphere* ptrTabSphere){
    Device::malloc(&ptrDevTabSphere, sizeOctet);
    Device::memclear(ptrDevTabSphere, sizeOctet);
    Device::memcpyHToD(ptrDevTabSphere, ptrTabSphere, sizeOctet);

}

void Raytracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath){
    //Device::lastCudaError("Before raytracing");
    raytracing<<<dg, db>>>(ptrDevPixels, w, h, t, ptrDevTabSphere, nbSphere);
    //Device::lastCudaError("After raytracing");

    //Ajouter barriere synchro implicite
}



Raytracing::~Raytracing(){
    Device::free(ptrDevTabSphere);
}

void Raytracing::animationStep(){
    t += dt;
}

