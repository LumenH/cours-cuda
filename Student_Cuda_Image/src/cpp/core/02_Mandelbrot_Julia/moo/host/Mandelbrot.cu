#include "hip/hip_runtime.h"
#include "Mandelbrot.h"
#include "MandelbrotMath.h"
#include "DomaineMath_GPU.h"

#include <iostream>
#include <assert.h>
#include "Device.h"

using std::cout;
using std::endl;
/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/
extern __global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, float t, const uint n, DomaineMath domaineMath);
//extern __device__ void workPixel(uchar4* ptrColorIJ, int i, int j, const DomaineMath& domaineMath, MandelbrotMath* ptrMandelbrotMath);
/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Mandelbrot::Mandelbrot(const Grid& grid, uint w, uint h, int dt, uint n, const DomaineMath& domaineMath) :
	Animable_I<uchar4>(grid, w, h, "Damier_OMP_rgba_uchar4",domaineMath), variateurAnimation(Interval<int>(30, n), dt)
{
	this->n = n;
	this->t = 0;
}

Mandelbrot::~Mandelbrot(){

}


void Mandelbrot::animationStep(){
    this->t = variateurAnimation.varierAndGet();
}

void Mandelbrot::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath){
    Device::lastCudaError("Mandelbrot before kernel");

    mandelbrot<<<dg,db>>>(ptrDevPixels, w, h, t, n, domaineMath );

    Device::lastCudaError("Mandelbrot after kernel");
}
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

