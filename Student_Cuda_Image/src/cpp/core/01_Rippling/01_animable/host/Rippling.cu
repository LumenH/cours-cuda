#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Device.h"
#include "Rippling.h"
#include <assert.h>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void rippling(uchar4* ptrDevPixels,uint w, uint h,float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Rippling::Rippling(const Grid& grid, uint w, uint h, float dt) :
	Animable_I<uchar4>(grid, w, h, "Rippling_Cuda_RGBA_uchar4")
    {
    assert(w == h); // specific rippling

    // Inputs
    this->dt = dt;

    // Tools
    this->t = 0; // protected dans Animable
    }

Rippling::~Rippling()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Rippling::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("rippling rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    // TODO lancer le kernel avec <<<dg,db>>>
    // le kernel est importer ci-dessus (ligne 19)
<<<<<<< HEAD
=======
    rippling<<<dg,db>>>(ptrDevPixels, w, h, t);
>>>>>>> 1e91cebe3b4b949c7e3d13a3e5f185d4131eae5d

    Device::lastCudaError("rippling rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void Rippling::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

