#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"

#include "RipplingMath.h"
using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
<<<<<<< HEAD
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!
=======
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!
>>>>>>> 1e91cebe3b4b949c7e3d13a3e5f185d4131eae5d

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, uint w, uint h, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, uint w, uint h, float t)
    {
    RipplingMath ripplingMath = RipplingMath(w, h);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    // TODO pattern entrelacement
<<<<<<< HEAD
=======
    //Rippling.cpp OMP
    int s = TID;
    int i, j;
    while(s < WH){
    	 IndiceTools::toIJ(s, w, &i, &j);

    	    ripplingMath.colorIJ(&ptrDevPixels[s], i, j, t);

    	    s += NB_THREAD;
    	}
>>>>>>> 1e91cebe3b4b949c7e3d13a3e5f185d4131eae5d
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

